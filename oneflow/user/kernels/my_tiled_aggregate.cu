#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/common/data_type.h"
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/util/cuda_half_util.h"
#include "oneflow/core/ep/cuda/hip_stream.h"

namespace oneflow {

//namespace {

template<typename T>
__global__ void TiledAggregateGpu(
    const T *matrix_W, 
    const T *matrix_H, 
    const T *vector_b,
    const int tile_size,
    T *c,
    const int N) {
  // Compute each thread's global row and column index
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Statically allocated shared memory
  extern __shared__ int tile[];
  int* tile_W = tile;
  int* tile_H = tile+tile_size*tile_size;

  // Accumulate in temporary variable
  int tmp = 0;

  // Sweep tile across matrix
  for (int i = 0; i < N; i += blockDim.x) {
    // Load in elements for this tile
    int index_W = row * N + i + threadIdx.x;
    if(index_W < N*N){
      tile_W[threadIdx.y * blockDim.x + threadIdx.x] = matrix_W[index_W];
    }else{
      tile_W[threadIdx.y * blockDim.x + threadIdx.x] = 0;
    }
    int index_H = i * N + threadIdx.y * N + col;
    if(index_H < N*N){
      tile_H[threadIdx.y * blockDim.x + threadIdx.x] = matrix_H[index_H];
    }else{
      tile_H[threadIdx.y * blockDim.x + threadIdx.x] = 0;
    }
    __syncthreads();

    // Do matrix multiplication on the small matrix
    for (int j = 0; j < blockDim.x; j++) {
      tmp += tile_W[threadIdx.y * blockDim.x + j] * tile_H[j * blockDim.x + threadIdx.x];
    }
    __syncthreads();
  }

  // Write back results
  if(row < N && col < N){
    c[row * N + col] = vector_b[col] + tmp;
  }
}

//}  // namespace

template<typename T>
class GpuMyTiledAggregateKernel final : public user_op::OpKernel {
 public:
  GpuMyTiledAggregateKernel() = default;
  ~GpuMyTiledAggregateKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    //传入Tensor4ArgNameAndIndex的字符串要和之前在OneFlowUserOps.td设置的名称一致
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const user_op::Tensor* w = ctx->Tensor4ArgNameAndIndex("w", 0);
    const user_op::Tensor* b = ctx->Tensor4ArgNameAndIndex("b", 0);
    user_op::Tensor* y = ctx->Tensor4ArgNameAndIndex("y", 0);

    CHECK_EQ(x->shape_view().NumAxes(), 2) << "x Numdims should be equal to 2. ";
    const DataType data_type = x->data_type();
    CHECK_EQ(w->shape_view().NumAxes(), 2) << "w Numdims should be equal to 2. ";
    CHECK_EQ(x->data_type(), data_type) << "Matrix X Datatype should be equal to Vector b";
    CHECK_EQ(b->shape_view().NumAxes(), 1) << "b Numdims should be equal to 1. ";
    CHECK_EQ(b->data_type(), data_type) << "Matrix X Datatype should be equal to vector b";

    CHECK_EQ(y->shape_view().NumAxes(), 2) << "y Numdims should be equal to 2. ";
    CHECK_EQ(y->data_type(), data_type) << "y Datatype should be equal to input's. ";

    int N = x->shape_view().At(0);  //x，w，y矩阵的size都是N*N
    // Threads per CTA dimension
    int threads_per_block = 32;
    // Blocks per grid dimension 
    int blocks_num = ( N + threads_per_block -1 ) / threads_per_block;
    // Use dim3 structs for block  and grid dimensions
    dim3 threads(threads_per_block, threads_per_block);
    dim3 blocks(blocks_num, blocks_num);
    /*obtain shared memory size for each thread block(tile_A+tile_B,所以乘2)
      if threads_per_block=32, then shared_memory_size = 2*32*32*4 = 8192 bytes = 8 KB
      if threads_per_block=128, then shared_memory_size = 2*128*128*4 = 131072 bytes = 128 KB
      而GPU02上的shared memory size 是 49152 bytes = 48 KB*/
    int shared_memory_size = 2*threads_per_block*threads_per_block*sizeof(int);
    //launch kernel
    TiledAggregateGpu<T><<<blocks, threads, shared_memory_size>>>
                    (w->dptr<T>(), x->dptr<T>(), b->dptr<T>(), threads_per_block, y->mut_dptr<T>(), N);
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_GPU_MY_TILED_AGGREGATE_KERNEL(dtype)                  \
  REGISTER_USER_KERNEL("my_tiled_aggregate")                           \
      .SetCreateFn<GpuMyTiledAggregateKernel<dtype>>()                 \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("y", 0) == GetDataType<dtype>::value));

REGISTER_GPU_MY_TILED_AGGREGATE_KERNEL(float)
REGISTER_GPU_MY_TILED_AGGREGATE_KERNEL(double)
REGISTER_GPU_MY_TILED_AGGREGATE_KERNEL(uint8_t)
REGISTER_GPU_MY_TILED_AGGREGATE_KERNEL(int8_t)
REGISTER_GPU_MY_TILED_AGGREGATE_KERNEL(int32_t)
REGISTER_GPU_MY_TILED_AGGREGATE_KERNEL(int64_t)

}  // namespace oneflow